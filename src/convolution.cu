#include "hip/hip_runtime.h"
//#include <opencv2/gpu/gpu.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/version.hpp>
using namespace cv;

#if CV_VERSION_EPOCH == 2
#define OPENCV2
#include <opencv2/gpu/gpu.hpp>
namespace GPU = cv::gpu;

#elif CV_VERSION_MAJOR == 4 
#define  OPENCV4
#include <opencv2/core/cuda.hpp>
namespace GPU = cv::cuda;
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/imgproc.hpp>
#include <iostream>

#define THREAD_X 32
#define THREAD_Y 32
#define WRAP_SIZE 32
#define MAX_WRAP_NUM 32
#define KERNEL_SIZE 3

//using namespace cv;
//using namespace cv;
__global__ void conv(int* dev){
        int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
		dev[pixel_i]=1;
		//        printf("idx%d %d,%d\n",pixel_i,pixel_j,dev[pixel_i]);

}
__global__ void convolution(GPU::PtrStepSz<float1> src,GPU::PtrStepSz<double> guass_kernel,GPU::PtrStepSz<float1> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ int  share_mem[WRAP_SIZE][MAX_WRAP_NUM];
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
//	printf("idx%d %d\n",pixel_i,pixel_j);
	//need to do bound check
	//printf("pixel %d %d block dim %d %d\n",pixel_i,pixel_j,blockDim.x,blockDim.y);
	/*int thread_block_index=pixel_i+pixel_j*;
	int share_i=thread_block_index%WRAP_NUM;
	int share_j=thread_block_index/WRAP_NUM;*/
	float sum=0;
	//share_mem[share_i][share_j]=src(pixel_i,pixel_j);
	//share_mem[threadIdx.x][threadIdx.y]=src(pixel_i,pixel_j).x;
	__syncthreads();
	 //printf("%d %d %d\n",pixel_i,pixel_j,share_mem[pixel_i][pixel_j]);
	//printf("%lf\n",guass_kernel(0,0));
	if(!(pixel_i<kernel_radius || pixel_j<kernel_radius || pixel_i>=orign_width+kernel_radius  || pixel_j>=orign_height+kernel_radius)){
//		printf("blockidx.x %dblockidx.y %d x %d,y %d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
		int start_i=pixel_i-kernel_radius,start_j=pixel_j-kernel_radius;
		for(int i=0;i<kernel_size;i++){
			for(int j=0;j<kernel_size;j++){
				int index_i=start_i+i,index_j=start_j+j;
				//sum+=share_mem[][index_j]*guass_kernel(i,j).x;
				sum+=src(index_j,index_i).x*(float)guass_kernel(i,j);
			}
		}

		dst(pixel_j-kernel_radius,pixel_i-kernel_radius).x=sum;//sum;
		//dst(pixel_j-kernel_radius,pixel_i-kernel_radius).y=sum;
		//dst(pixel_j-kernel_radius,pixel_i-kernel_radius).z=sum;
		//dst(pixel_j-kernel_radius,pixel_i-kernel_radius)=src(pixel_j-kernel_radius,pixel_i-kernel_radius);
//		printf("i:%d j:%d value:%d\n",pixel_i-kernel_radius,pixel_j-kernel_radius,dst(pixel_j-kernel_radius,pixel_i-kernel_radius).x);
	}
	return ;
}

void guassain_conv(const Mat *src,Mat *dst,double sigma){
	int kernel_radius=KERNEL_SIZE/2;
	int orign_width=src->cols,orign_height=src->rows;
	Mat padding_image;
	GPU::GpuMat device_image,g_kernel,result;
	//*dst=Mat(src->rows,src->cols,0);	

	if(GPU::getCudaEnabledDeviceCount()==0){
		std::cout<<"not use GPU module"<<std::endl;
		return ;
	}
	std::cout<<dst->type()<<std::endl;	
	std::cout<<dst->cols<<"rows"<<dst->rows<<std::endl;
	Mat gauss_x=getGaussianKernel(KERNEL_SIZE,sigma),gauss_y=getGaussianKernel(KERNEL_SIZE,sigma); //3*3 filter
	Mat gauss_kernel=gauss_x*gauss_y.t();
	std::cout<<gauss_kernel<<std::endl;
	copyMakeBorder(*src,padding_image,kernel_radius,kernel_radius,kernel_radius,kernel_radius,BORDER_CONSTANT, 0);
	//printf("%d %d %d %d\n",src->rows,src->cols,padding_image.rows,padding_image.cols);
	int grid_num_x=(padding_image.cols+THREAD_X-1)/THREAD_X,grid_num_y=(padding_image.rows+THREAD_Y-1)/THREAD_Y;
	//printf("%d %d\n",grid_num_x,grid_num_y);
	result.upload(*dst);
	int *h,*dev;
	printf("%d %d %d\n",gauss_kernel.rows,gauss_kernel.cols,gauss_kernel.channels());
	g_kernel.upload(gauss_kernel);
	device_image.upload(padding_image);
	dim3 thread_block(THREAD_X,THREAD_Y);
	dim3 grid(grid_num_x,grid_num_y);
	convolution<<<grid,thread_block>>>(device_image,g_kernel,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);

	Mat re;
	result.download(re);
	//std::cout<<src->type()<<re.type()<<std::endl;
	//std::cout<<(*src-re)<<std::endl;
	printf("%d %d %d %d %d %d\n",src->rows,src->cols,src->channels(),re.rows,re.cols,re.channels());
	*dst=re.clone();
	imwrite("re.png",re);
	return ;
}


