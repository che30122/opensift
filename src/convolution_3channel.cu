#include "hip/hip_runtime.h"
//#include <opencv2/gpu/gpu.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/version.hpp>
#include <opencv2/imgproc/imgproc.hpp>
using namespace cv;

#if CV_VERSION_EPOCH == 2
#define OPENCV2
#include <opencv2/gpu/gpu.hpp>
namespace GPU = cv::gpu;
#elif CV_VERSION_MAJOR == 4 
#define  OPENCV4
#include <opencv2/core/cuda.hpp>
namespace GPU = cv::cuda;
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/imgproc.hpp>
#include <iostream>

#define THREAD_X 32
#define THREAD_Y 32
#define THREAD_X1 32
#define THREAD_Y1 32
#define WRAP_NUM 32
#define MAX_WRAP_NUM 32

//using namespace cv;
//using namespace cv;

__constant__ double guass_kernel[2048];
__constant__ double guass_kernel_x[128*2];
__constant__ double guass_kernel_y[128];
static int KERNEL_SIZE;

//not need to padding
__global__ void conv_3x(GPU::PtrStepSz<uchar3> src,/*const double* __restrict__ guass_kernel,*/GPU::PtrStepSz<uchar3> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ float  share_mem[100][100];
	int left_limit=kernel_radius,right_limit=blockDim.x-kernel_radius;
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x-2*blockIdx.x*kernel_radius;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y;
	int thread_block_index=threadIdx.x+threadIdx.y*blockDim.x;
//	share_mem[thread_block_index%32][thread_block_index/32]=src(pixel_j,pixel_i);
	//share_mem[10]=src(pixel_j,pixel_i);
	__syncthreads();
	float sum=0,sum1=0,sum2=0;
	if(!(pixel_i<kernel_radius || pixel_j<kernel_radius || pixel_i>=orign_width+kernel_radius  || pixel_j>=orign_height+kernel_radius)){//real image size
		if(threadIdx.x>= left_limit && threadIdx.x<right_limit){ //non padding size
			int x=threadIdx.x-kernel_radius,y=threadIdx.y;

			for(int i=0;i<kernel_size;i++){
				
				thread_block_index=(x+i)+y*blockDim.x;
	//			if(thread_block_index>=2048 || thread_block_index<0)
	//				                                        printf("%d\n",thread_block_index/WRAP_NUM);
				sum+=src(pixel_j,pixel_i-kernel_radius+i).x*(float)guass_kernel_x[i];
	//			sum+=share_mem[thread_block_index%32][thread_block_index/32]*(float)guass_kernel_x[i];
				sum1+=src(pixel_j,pixel_i-kernel_radius+i).y*(float)guass_kernel_x[i];
				sum2+=src(pixel_j,pixel_i-kernel_radius+i).z*(float)guass_kernel_x[i];
			}
			dst(pixel_j-kernel_radius,pixel_i-kernel_radius).x=sum;//sum;//src(pixel_j,pixel_i);
			dst(pixel_j-kernel_radius,pixel_i-kernel_radius).y=sum1;//sum1;
			dst(pixel_j-kernel_radius,pixel_i-kernel_radius).z=sum2;//sum2;
		}
		//dst(pixel_j,pixel_i)=sum;	
	}
	//dst(pixel_j,pixel_i)=sum;
	return ;
}
__global__ void conv_3y(GPU::PtrStepSz<uchar3> src,/*const double* __restrict__ guass_kernel,*/GPU::PtrStepSz<uchar3> dst,int kernel_size,int kernel_radius,int orign_width,int orign_height){
	__shared__ float  share_mem[100][100];
	int top_limit=kernel_radius,down_limit=blockDim.y-kernel_radius;
	int pixel_i=blockDim.x*blockIdx.x+threadIdx.x;
	int pixel_j=blockDim.y*blockIdx.y+threadIdx.y-2*blockIdx.y*kernel_radius;
	int thread_block_index=threadIdx.x+threadIdx.y*blockDim.x;
//	share_mem[thread_block_index%32][thread_block_index/32]=src(pixel_j,pixel_i);
	__syncthreads();
	float sum=0.0,sum1=0,sum2=0;

	if(!(pixel_i<kernel_radius || pixel_j<kernel_radius || pixel_i>=orign_width+kernel_radius  || pixel_j>=orign_height+kernel_radius)){
		if(threadIdx.y>= top_limit && threadIdx.y<down_limit){
			int x=threadIdx.x,y=threadIdx.y-kernel_radius;
			for(int i=0;i<kernel_size;i++){
				thread_block_index=x+(y+i)*blockDim.x;
//				sum+=share_mem[thread_block_index%32][thread_block_index/32]*(float)guass_kernel_x[i];
				sum+=src(pixel_j-kernel_radius+i,pixel_i).x*(float)guass_kernel_x[i];
				
				sum1+=/*share_mem[thread_block_index%WRAP_NUM][thread_block_index/WRAP_NUM]*/src(pixel_j-kernel_radius+i,pixel_i).y*(float)guass_kernel_x[i];
				                                sum2+=src(pixel_j-kernel_radius+i,pixel_i).z*(float)guass_kernel_x[i];
			}
		
		dst(pixel_j-kernel_radius,pixel_i-kernel_radius).x=sum;//sum;//src(pixel_j,pixel_i);//sum;
		dst(pixel_j-kernel_radius,pixel_i-kernel_radius).y=sum1;//sum1;
		                        dst(pixel_j-kernel_radius,pixel_i-kernel_radius).z=sum2;//sum2;
		}
	}
	//dst(pixel_j,pixel_i)=sum;
	return ;
}

void guassain_3conv(const Mat *src,Mat *dst,double sigma){
//	int depth = CV_MAT_DEPTH(src.type());
	KERNEL_SIZE = cvRound(sigma* 4 * 2 + 1)|1;
	std::cout<<KERNEL_SIZE<<std::endl;
	int kernel_radius=KERNEL_SIZE/2;
	int orign_width=src->cols,orign_height=src->rows;
	Mat padding_image;
	GPU::GpuMat device_image,g_kernel,result, dev_image,resul;

	if(GPU::getCudaEnabledDeviceCount()==0){
		std::cout<<"not use GPU module"<<std::endl;
		return ;
	}
	Mat gauss_x=getGaussianKernel(KERNEL_SIZE,sigma),gauss_y=getGaussianKernel(KERNEL_SIZE,sigma); //3*3 filter
	//Mat gauss_kernel=gauss_x*gauss_y.t();
	//allocate
	/*double* gs_kernel,*dev_kernel;
	hipHostAlloc(&gs_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE,hipHostMallocDefault);
	for(int i=0;i<KERNEL_SIZE;i++){
		double* row=gauss_kernel.ptr<double>(i);
		for(int j=0;j<KERNEL_SIZE;j++){
			gs_kernel[i*KERNEL_SIZE+j]=row[j];
		}
	}*/
	//hipMalloc(&dev_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE);
	//allocate 
	//allocate
	double* x,*y;
	hipHostAlloc(&x,sizeof(double)*KERNEL_SIZE*2,hipHostMallocDefault);
	double *row_x=gauss_x.ptr<double>(0),*row_y=gauss_y.ptr<double>(0);
	for(int i=0;i<KERNEL_SIZE*2;i++){
		if(i<KERNEL_SIZE){
			x[i]=row_x[i];
			//std::cout<<x[i]<<std::endl;
		}
		else
			x[i]=row_y[i-KERNEL_SIZE];
	}
	//hipHostAlloc(&y,sizeof(double)*KERNEL_SIZE,hipHostMallocDefault);
	//allocate
	copyMakeBorder(*src,padding_image,kernel_radius,kernel_radius,kernel_radius,kernel_radius,BORDER_CONSTANT, 0);
	//int orign_grid_num_x=(padding_image.cols+THREAD_X-1)/THREAD_X,orign_grid_num_y=(padding_image.rows+THREAD_Y-1)/THREAD_Y;
	int t_x=THREAD_X-2*kernel_radius,t_y=THREAD_Y;
	int grid_num_x=(padding_image.cols+t_x-1)/t_x,grid_num_y=(padding_image.rows+t_y-1)/t_y;
	//int grid_num_x=orign_grid_num_x+(2*kernel_radius*orign_grid_num_x+THREAD_X-1)/THREAD_X,grid_num_y=orign_grid_num_y+(2*kernel_radius*orign_grid_num_y+THREAD_Y-1)/THREAD_Y;
	//int grid_num_x=(src->cols+THREAD_X-1)/THREAD_X,grid_num_y=(src->rows+THREAD_Y-1)/THREAD_Y;
	result.upload(*dst);
	//g_kernel.upload(gauss_kernel);

	//use seperate do no padding
	//device_image.upload(padding_image);

	device_image.upload(padding_image);
	//device_image.upload(*src);
	hipMemcpyToSymbol(HIP_SYMBOL(guass_kernel_x),x,sizeof(double)*2*KERNEL_SIZE);
	//hipMemcpyToSymbol(HIP_SYMBOL(guass_kernel),gs_kernel,sizeof(double)*KERNEL_SIZE*KERNEL_SIZE);
	dim3 thread_block(THREAD_X,THREAD_Y);
	dim3 grid(grid_num_x,grid_num_y);
	//convolution<<<grid,thread_block>>>(device_image,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	conv_3x<<<grid,thread_block>>>(device_image,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	hipDeviceSynchronize();
	Mat re;
	result.download(re);
	copyMakeBorder(re,padding_image,kernel_radius,kernel_radius,kernel_radius,kernel_radius,BORDER_CONSTANT, 0);
	//resul.upload(re);
	device_image.upload(padding_image);  
	

	
	//orign_grid_num_x=(padding_image.cols+THREAD_X1-1)/THREAD_X1,orign_grid_num_y=(padding_image.rows+THREAD_Y1-1)/THREAD_Y1;
	t_x=THREAD_X1;
	t_y=THREAD_Y1-2*kernel_radius;
        //grid_num_x=orign_grid_num_x+(2*kernel_radius*orign_grid_num_x+THREAD_X1-1)/THREAD_X1,grid_num_y=orign_grid_num_y+(2*kernel_radius*orign_grid_num_y+THREAD_Y1-1)/THREAD_Y1;
	grid_num_x=(padding_image.cols+t_x-1)/t_x,grid_num_y=(padding_image.rows+t_y-1)/t_y;
	dim3 thread_block1(THREAD_X1,THREAD_Y1);
        dim3 grid1(grid_num_x,grid_num_y);
	conv_3y<<<grid1,thread_block1>>>(device_image,result,KERNEL_SIZE,kernel_radius,orign_width,orign_height);
	result.download(*dst);
	return ;
}


